#include <iostream>
#include <hip/hip_runtime.h>

#define N 10000  // Matrix size (N x N)

// CUDA kernel for simple matrix addition
__global__ void matrixAdd(float* A, float* B, float* C, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < n && col < n) {
        C[row * n + col] = A[row * n + col] + B[row * n + col];
    }
}

int main() {
    int size = N * N * sizeof(float);

    // Host matrices
    float* h_A = (float*)malloc(size);
    float* h_B = (float*)malloc(size);
    float* h_C = (float*)malloc(size);

    // Initialize matrices A and B
    for (int i = 0; i < N * N; i++) {
        h_A[i] = rand() / (float)RAND_MAX;
        h_B[i] = rand() / (float)RAND_MAX;
    }

    // Device matrices
    float *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    // Copy matrices from host to device
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Define block and grid sizes
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((N + threadsPerBlock.x - 1) / threadsPerBlock.x, 
                       (N + threadsPerBlock.y - 1) / threadsPerBlock.y);

    // Timing events
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Record start event
    hipEventRecord(start);

    // Launch the matrix addition kernel
    matrixAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    // Record stop event
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // Calculate elapsed time
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    // Copy the result back to host
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Output the time taken
    std::cout << "Time taken for matrix addition on GPU: " << milliseconds << " ms" << std::endl;

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}